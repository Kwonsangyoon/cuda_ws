#include <hip/hip_runtime.h>

#include <stdio.h>
#include <assert.h>

__global__ void hellocuda()
{
    printf("cuda test!\n");
}

int main(void)
{
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    printf("Running on GPU %d: %s\n", device, props.name);

    hellocuda<<<10, 1>>>();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error after kernel launch: %s\n", hipGetErrorString(error));
        return -1;
    }

    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error after synchronization: %s\n", hipGetErrorString(error));
        return -1;
    }

    return 0;
}
