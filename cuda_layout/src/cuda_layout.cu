#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void checkIndex(void)
{
    printf("thread Idx:(%d, %d, %d)\n blockIdx: (%d, %d, %d)\n blockDim:(%d, %d, %d)\n Grid:(%d, %d, %d)\n",
    threadIdx.x,threadIdx.y,threadIdx.z,
    blockIdx.x,blockIdx.y,blockIdx.z,
    blockDim.x,blockDim.y,blockDim.z,
    gridDim.x,gridDim.y,gridDim.z
);
}

int main(){
    dim3 dimblock(2,1,16);
    dim3 dimgrid(1,1,1);

    checkIndex<<<dimgrid,dimblock>>>();
    hipDeviceSynchronize(); 
    return 0;
}