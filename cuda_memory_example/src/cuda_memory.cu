#include "../include/cuda_memory.h"

int main(void)
{
    int* dDataPtr;
    hipError_t errorCode;
    checkDeviceMemory();
    errorCode = hipMalloc(&dDataPtr,sizeof(int) * 1024 * 1024);
    printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

    errorCode = hipMemset( dDataPtr, 0, sizeof(int) * 1024 * 1024);
    printf("hipMemset - %s\n", hipGetErrorName(errorCode));

    errorCode = hipFree( dDataPtr);
    printf("hipFree - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();


    return 0;
}