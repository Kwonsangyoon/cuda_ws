#include "hip/hip_runtime.h"
#include "timer_test.cuh"
#include <chrono>

__global__ void vecAdd(int* _a, int* _b, int* _c)
{
    int tID = threadIdx.x;
    _c[tID] = _a[tID] + _b[tID];
}


int main()
{
    int *a, *b, *c, *hc;
    int *da, *db, *dc;
    


    int memSize = sizeof(int) * NUM_DATA;

    a = new int[memSize];
    b = new int[memSize];
    c = new int[memSize];
    hc = new int[memSize];
    memset(a,0x00,memSize);
    memset(b,0x00,memSize);
    memset(c,0x00,memSize);
    memset(hc,0x00,memSize);


    for(int i = 0 ; i < NUM_DATA ; i ++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
        hc[i] = a[i] + b[i];
    }

    hipMalloc(&da,memSize);
    hipMalloc(&db,memSize);
    hipMalloc(&dc,memSize);

    hipMemset(da,0x00,memSize);
    hipMemset(db,0x00,memSize);
    hipMemset(dc,0x00,memSize);


    hipMemcpy(da,a,memSize,hipMemcpyHostToDevice);
    hipMemcpy(db,b,memSize,hipMemcpyHostToDevice);
    std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
    vecAdd<<<1,NUM_DATA>>>(da,db,dc);
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
    std::chrono::duration<long long, std::micro> microsec = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    printf("take time : %lld\n", microsec);
    hipMemcpy(c,dc,memSize,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    
    bool result = true;

    for(int i = 0; i < NUM_DATA; i++)
    {
        if(hc[i] != c[i])
        {
            printf("[%d] The result is not matched (%d, %d)\n", i, hc[i], c[i]);
            result = false;
        }
    }
    if(result)
    {printf("GPU work well\n");}

    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}