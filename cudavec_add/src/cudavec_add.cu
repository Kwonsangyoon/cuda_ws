#include "hip/hip_runtime.h"
#include "cudavec_add.cuh"

__global__ void vecAdd(int* _a, int* _b, int* _c)
{
    int tID = threadIdx.x;
    _c[tID] = _a[tID] + _b[tID];
}


int main()
{
    int *a, *b, *c, *hc;
    int *da, *db, *dc;


    int memSize = sizeof(int) * NUM_DATA;

    a = new int[memSize];
    b = new int[memSize];
    c = new int[memSize];
    hc = new int[memSize];
    memset(a,0x00,memSize);
    memset(b,0x00,memSize);
    memset(c,0x00,memSize);
    memset(hc,0x00,memSize);


    for(int i = 0 ; i < NUM_DATA ; i ++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
        hc[i] = a[i] + b[i];
    }

    hipMalloc(&da,memSize);
    hipMalloc(&db,memSize);
    hipMalloc(&dc,memSize);

    hipMemset(da,0x00,memSize);
    hipMemset(db,0x00,memSize);
    hipMemset(dc,0x00,memSize);


    hipMemcpy(da,a,memSize,hipMemcpyHostToDevice);
    hipMemcpy(db,b,memSize,hipMemcpyHostToDevice);

    vecAdd<<<1,NUM_DATA>>>(da,db,dc);

    hipMemcpy(c,dc,memSize,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    
    bool result = true;

    for(int i = 0; i < NUM_DATA; i++)
    {
        if(hc[i] != c[i])
        {
            printf("[%d] The result is not matched (%d, %d)\n", i, hc[i], c[i]);
            result = false;
        }
    }
    if(result)
    {printf("GPU work well\n");}

    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}